/*
Lessons: 
1. Generally all of the kernel calls in CUDA are async to allow both CPU and GPU to work concurrently. 
2. cudaDeviceSynchronize() blocks the CPU to wait till GPU has completed execution. Thus, print_from_gpu is always
visibile when it is used. 
3. When sync() call is removed, the GPU print might not be visible. 
*/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu(){
    printf("Hello from GPU, thread[%d, %d]\n", threadIdx.x, blockIdx.x);
}

int main(){
    
    printf("Hello from HOST! \n");
    print_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}